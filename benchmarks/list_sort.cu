
#include <hip/hip_runtime.h>
#include <list>

#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cassert>

#define ITERATIONS 5

using namespace std;

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  list<int> int_list;

  unsigned long N = 32000;

  if(argc > 1){
    N = atol(argv[1]);
  }

  for(int i = 0; i < ITERATIONS; i++){

    for(int c = N; c > 0; c--){
      int_list.push_back(rand() % 500000);
    }

    clock_gettime(CLOCK_REALTIME,&start);

    for(int c = 0; c < ITERATIONS; c++){
      int_list.sort();
    }

    clock_gettime(CLOCK_REALTIME,&stop);
   
    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);

    int_list.clear();
  }

  int val = 0;
  for(std::list<int>::iterator it = int_list.begin(), e = int_list.end(); it != e; ++it){
    assert(*it >= val);
    val = *it;
  }

  printf("%lu ns\n", total_time);

  return 0;
}
