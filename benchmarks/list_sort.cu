
#include <hip/hip_runtime.h>
#include <list>

#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cassert>

#define ITERATIONS 5

#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

using namespace std;

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  list<DATA_TYPE> int_list;

  unsigned long N = 32000;

  if(argc > 1){
    N = atol(argv[1]);
  }

  for(int i = 0; i < ITERATIONS; i++){

    for(int c = N; c > 0; c--){
      int_list.push_back((DATA_TYPE)(rand() % 500000));
    }

    clock_gettime(CLOCK_REALTIME,&start);

    for(int c = 0; c < ITERATIONS; c++){
      int_list.sort();
    }

    clock_gettime(CLOCK_REALTIME,&stop);
   
    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);

    int_list.clear();
  }

  int val = 0;
  for(std::list<DATA_TYPE>::iterator it = int_list.begin(), e = int_list.end(); it != e; ++it){
    assert(*it >= val);
    val = *it;
  }

  printf("%lu\n", total_time);

  return 0;
}
