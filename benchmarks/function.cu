
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#include <cstdlib>
#include <cstdio>
#include <unistd.h>
#include <time.h>

#include <cassert>

#define ITERATIONS 5

#define M 50

using namespace std;

#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

struct Foo{
  __host__ __device__ DATA_TYPE operator() (DATA_TYPE i){
    for(int c = 0; c < M; c++){
      i += 5;
      //i /= 3;
      //i += 2;
      //i *= 11;
      //i /= 7;
      //i *= 10;
      //i += 2;
    }

    return i; 
  }
} myobject;

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  vector<DATA_TYPE> int_vector;

  DATA_TYPE lastVal = 0;

  unsigned long N = 32000;

  if(argc > 1){
    N = atol(argv[1]);
  }

  for(int i = 0; i < ITERATIONS; i++){
    int_vector.clear();

    for(int c = N; c > 0; c--){
      int_vector.push_back((DATA_TYPE)(rand() % 500000));
    }

    lastVal = int_vector[0];

    clock_gettime(CLOCK_REALTIME,&start);

    for(int c = 0; c < ITERATIONS; c++){
      std::transform(int_vector.begin(),int_vector.end(),int_vector.begin(),myobject);
    }

    clock_gettime(CLOCK_REALTIME,&stop);

    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);
  }

  assert(lastVal != int_vector[0]);

  printf("%lu\n", total_time);

  return 0;
}
