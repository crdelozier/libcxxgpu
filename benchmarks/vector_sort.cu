
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cassert>

#define ITERATIONS 5

using namespace std;

#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  vector<DATA_TYPE> int_vector;

  unsigned long N = 32000;

  if(argc > 1){
    N = atol(argv[1]);
  }

  for(int i = 0; i < ITERATIONS; i++){

    for(int c = N; c > 0; c--){
      int_vector.push_back((DATA_TYPE)(rand() % 500000));
    }

    clock_gettime(CLOCK_REALTIME,&start);

    for(int c = 0; c < ITERATIONS; c++){
      sort(int_vector.begin(),int_vector.end());
    }

    clock_gettime(CLOCK_REALTIME,&stop);
   
    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);

    int_vector.clear();
  }

  int val = 0;
  for(std::vector<DATA_TYPE>::iterator it = int_vector.begin(), e = int_vector.end(); it != e; ++it){
    assert(*it >= val);
    val = *it;
  }

  printf("%lu\n", total_time);

  return 0;
}
