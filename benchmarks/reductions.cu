
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cassert>

#define ITERATIONS 5

using namespace std;

#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  vector<DATA_TYPE> int_vector;

  unsigned long N = 32000;

  DATA_TYPE sum1 = 0;
  DATA_TYPE sum2 = 0;

  if(argc > 1){
    N = atol(argv[1]);
  }

  for(int i = 0; i < ITERATIONS; i++){

    for(int c = N; c > 0; c--){
      int_vector.push_back((DATA_TYPE)(rand() % 500000));
    }

    clock_gettime(CLOCK_REALTIME,&start);

    for(int c = 0; c < ITERATIONS; c++){
      sum1 += *min_element(int_vector.begin(),int_vector.end());
      sum2 += *max_element(int_vector.begin(),int_vector.end());
    }

    clock_gettime(CLOCK_REALTIME,&stop);
   
    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);

    int_vector.clear();
  }

  assert(sum1 != sum2);

  printf("%lu\n", total_time);

  return 0;
}
