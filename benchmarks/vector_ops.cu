
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>

#include <cstdlib>
#include <cstdio>
#include <time.h>

#include <cassert>

#define ITERATIONS 1
#define FINDS 10000

#define M 1046527

#define LINEAR 1
#define BINARY 2

using namespace std;

int main(int argc, char ** argv){
  long total_time = 0;
  struct timespec start, stop;
  std::vector<int> vec;

  unsigned long N = 32000;
  int method = 1;

  if(argc > 1){
    N = atol(argv[1]);
    if(N <= 0){
      printf("N must be greater than 0\n");
      return 1;
    }
  }

  if(argc > 2){
    method = atoi(argv[2]);
    if(method != 1 && method != 2){
      printf("Search method parameters may be 1 (Linear) or 2 (Binary).\n");
      return 1;
    }
  }

  vec.resize(N);
  srand(100);
  unsigned long numFound = 0;

  for(int i = 0; i < ITERATIONS; i++){
    for(int c = 0; c < N; c++){
      vec[c] = rand() % M;
    }

    clock_gettime(CLOCK_REALTIME,&start);

    if(method == LINEAR){
      for(int c = 0; c < FINDS; c++){
        if(find(vec.begin(), vec.end(), rand() % M) != vec.end()){
	  ++numFound;
	}
      }
    }else{
      sort(vec.begin(),vec.end());
      for(int c = 0; c < FINDS; c++){
        if(binary_search(vec.begin(), vec.end(), rand() % M)){
	  ++numFound;
	}
      }
    }

    clock_gettime(CLOCK_REALTIME,&stop);
   
    total_time += ((stop.tv_sec-start.tv_sec)*1000000000) + (stop.tv_nsec - start.tv_nsec);
  }

  printf("%lu ns\n", total_time);

  printf("Found: %lu\n",numFound);

  return 0;
}
